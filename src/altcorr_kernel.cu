#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <THC/THCAtomics.cuh>
#include <vector>
#include <iostream>

using namespace torch::indexing;

#define THREADS 256
#define BLOCKS(n) (n + THREADS - 1) / THREADS

#ifdef _WIN32
    #include <cstdint>
    typedef int64_t LongType;
#else
    typedef long LongType;
#endif

__forceinline__ __device__
bool within_bounds(int h, int w, int H, int W) {
  return h >= 0 && h < H && w >= 0 && w < W;
}


template <typename scalar_t>
__global__ void corr_forward_kernel(int R,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> fmap1,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> fmap2,
    const torch::PackedTensorAccessor32<float,5,torch::RestrictPtrTraits> coords,
    const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> us,
    const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> vs,
    torch::PackedTensorAccessor32<scalar_t,6,torch::RestrictPtrTraits> corr)
{
  // diameter
  const int D = 2*R + 2;

  const int B = coords.size(0);
  const int M = coords.size(1);
  const int H = coords.size(3);
  const int W = coords.size(4);

  const int C = fmap1.size(2);
  const int H2 = fmap2.size(3);
  const int W2 = fmap2.size(4);

  int n = blockIdx.x * blockDim.x + threadIdx.x;

  if (n < B * M * H * W * D * D) {
    const int jj = n % D; n /= D;
    const int ii = n % D; n /= D;
    const int j0 = n % W; n /= W;
    const int i0 = n % H; n /= H;
    const int  m = n % M; n /= M;

    const int ix = us[m];
    const int jx = vs[m];

    const float x = coords[n][m][0][i0][j0];
    const float y = coords[n][m][1][i0][j0];

    const int i1 = static_cast<int>(floor(y)) + (ii - R);
    const int j1 = static_cast<int>(floor(x)) + (jj - R);

    // accumulate in fp32
    float s = 0;
    if (within_bounds(i1, j1, H2, W2)) {
      for (int i = 0; i < C; i++) {
        const scalar_t f1 = fmap1[n][ix][i][i0][j0] / 4.0;
        const scalar_t f2 = fmap2[n][jx][i][i1][j1] / 4.0;
        s += static_cast<float>(f1 * f2);
      }
    }

    corr[n][m][ii][jj][i0][j0] = static_cast<scalar_t>(s);
  }
}


template <typename scalar_t>
__global__ void corr_backward_kernel(int R,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> fmap1,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> fmap2,
    const torch::PackedTensorAccessor32<float,5,torch::RestrictPtrTraits> coords,
    const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> us,
    const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> vs,
    const torch::PackedTensorAccessor32<float,6,torch::RestrictPtrTraits> corr_grad,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> fmap1_grad,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> fmap2_grad)
{
  // diameter
  const int D = 2*R + 2;

  const int B = coords.size(0);
  const int M = coords.size(1);
  const int H = coords.size(3);
  const int W = coords.size(4);

  const int C = fmap1.size(2);
  const int H2 = fmap2.size(3);
  const int W2 = fmap2.size(4);

  int n = blockIdx.x * blockDim.x + threadIdx.x;

  if (n < B * M * H * W * D * D) {
    const int jj = n % D; n /= D;
    const int ii = n % D; n /= D;
    const int j0 = n % W; n /= W;
    const int i0 = n % H; n /= H;
    const int  m = n % M; n /= M;

    const int ix = us[m];
    const int jx = vs[m];

    const float x = coords[n][m][0][i0][j0];
    const float y = coords[n][m][1][i0][j0];

    const int i1 = static_cast<int>(floor(y)) + (ii - R);
    const int j1 = static_cast<int>(floor(x)) + (jj - R);

    const scalar_t g = (scalar_t) corr_grad[n][m][ii][jj][i0][j0];

    if (within_bounds(i1, j1, H2, W2)) {
      #pragma unroll 32
      for (int i=0; i<C; i++) {
        atomicAdd(&fmap1_grad[n][ix][i][i0][j0], g * fmap2[n][jx][i][i1][j1]);
        atomicAdd(&fmap2_grad[n][jx][i][i1][j1], g * fmap1[n][ix][i][i0][j0]);
      }
    }
  }
}


std::vector<torch::Tensor> altcorr_cuda_forward(
  torch::Tensor fmap1,
  torch::Tensor fmap2,
  torch::Tensor coords,
  torch::Tensor ii,
  torch::Tensor jj,
  int radius)
{
  const int B = coords.size(0);
  const int M = coords.size(1);

  const int H = coords.size(3);
  const int W = coords.size(4);
  const int D = 2 * radius + 2;

  auto opts = fmap1.options();
  auto corr = torch::empty({B, M, D, D, H, W}, opts);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(fmap1.scalar_type(), "corr_forward_kernel", ([&] {
      corr_forward_kernel<scalar_t><<<BLOCKS(B * M * H * W * D * D), THREADS>>>(radius,
        fmap1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        fmap2.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
        coords.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
        ii.packed_accessor32<LongType,1,torch::RestrictPtrTraits>(),
        jj.packed_accessor32<LongType,1,torch::RestrictPtrTraits>(),
        corr.packed_accessor32<scalar_t,6,torch::RestrictPtrTraits>());
  }));

  torch::Tensor x = coords.index({Slice(), Slice(), 0, None, None});
  torch::Tensor y = coords.index({Slice(), Slice(), 1, None, None});
  torch::Tensor dx = x - x.floor(); dx = dx.to(fmap1.dtype());
  torch::Tensor dy = y - y.floor(); dy = dy.to(fmap2.dtype());

  torch::Tensor out;
  out  = (1 - dx) * (1 - dy) * corr.index({Slice(), Slice(), Slice(0, D-1), Slice(0, D-1)});
  out +=     (dx) * (1 - dy) * corr.index({Slice(), Slice(), Slice(0, D-1), Slice(1, D-0)});
  out += (1 - dx) *     (dy) * corr.index({Slice(), Slice(), Slice(1, D-0), Slice(0, D-1)});
  out +=     (dx) *     (dy) * corr.index({Slice(), Slice(), Slice(1, D-0), Slice(1, D-0)});

  return { out.permute({0,1,3,2,4,5}) };
}


std::vector<torch::Tensor> altcorr_cuda_backward(
  torch::Tensor fmap1,
  torch::Tensor fmap2,
  torch::Tensor coords,
  torch::Tensor ii,
  torch::Tensor jj,
  torch::Tensor grad,
  int radius)
{
  const int B = coords.size(0);
  const int M = coords.size(1);

  const int H = coords.size(3);
  const int W = coords.size(4);
  const int D = 2 * radius + 2;
   
  grad = grad.permute({0,1,3,2,4,5}).contiguous();
  torch::Tensor x = coords.index({Slice(), Slice(), 0, None, None});
  torch::Tensor y = coords.index({Slice(), Slice(), 1, None, None});
  torch::Tensor dx = x - x.floor();
  torch::Tensor dy = y - y.floor();

  auto opts = torch::TensorOptions().dtype(torch::kFloat).device(torch::kCUDA);
  torch::Tensor g1 = torch::zeros({B, M, D, D, H, W}, grad.options());
  torch::Tensor g2 = torch::zeros({B, M, D, D, H, W}, grad.options());
  torch::Tensor g3 = torch::zeros({B, M, D, D, H, W}, grad.options());
  torch::Tensor g4 = torch::zeros({B, M, D, D, H, W}, grad.options());
  
  g1.index_put_({Slice(), Slice(), Slice(0, D-1), Slice(0, D-1)}, (1 - dx) * (1 - dy) * grad);
  g2.index_put_({Slice(), Slice(), Slice(0, D-1), Slice(1, D-0)},     (dx) * (1 - dy) * grad); 
  g3.index_put_({Slice(), Slice(), Slice(1, D-0), Slice(0, D-1)}, (1 - dx) *     (dy) * grad);
  g4.index_put_({Slice(), Slice(), Slice(1, D-0), Slice(1, D-0)},     (dx) *     (dy) * grad);

  torch::Tensor corr_grad = g1 + g2 + g3 + g4;
  auto fmap1_grad = torch::zeros_like(fmap1);
  auto fmap2_grad = torch::zeros_like(fmap2);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(fmap1.scalar_type(), "corr_backward_kernel", ([&] {
    corr_backward_kernel<scalar_t><<<BLOCKS(B * M * H * W * D * D), THREADS>>>(radius,
      fmap1.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
      fmap2.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
      coords.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
      ii.packed_accessor32<LongType,1,torch::RestrictPtrTraits>(),
      jj.packed_accessor32<LongType,1,torch::RestrictPtrTraits>(),
      corr_grad.packed_accessor32<float,6,torch::RestrictPtrTraits>(),
      fmap1_grad.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
      fmap2_grad.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>());
  }));

  return {fmap1_grad, fmap2_grad};
}
